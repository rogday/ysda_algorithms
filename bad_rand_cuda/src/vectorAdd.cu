#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <cstdint>

#include <chrono>
#include <iostream>

#include <hip/hip_runtime_api.h>

constexpr uint8_t TARGET[11] = "NLXGI4NoAp";
constexpr uint8_t ALPHABET[63] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
constexpr size_t FORWARD_BY = 100'000'000 + 310;
constexpr size_t STATE_SIZE = 31;
constexpr int64_t MODULO = 2'147'483'647ll;

__global__ void
bruteforce(const uint32_t* forward_matrix, const int64_t* pows, size_t begin)
{
    constexpr uint8_t TARGET[11] = "NLXGI4NoAp";
    constexpr uint8_t ALPHABET[63] = "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789";
    constexpr size_t FORWARD_BY = 100'000'000 + 310;
    constexpr size_t STATE_SIZE = 31;
    constexpr int64_t MODULO = 2'147'483'647ll;

    size_t threads = gridDim.x * gridDim.y * blockDim.x * blockDim.y;

    size_t blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    size_t index = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;

    uint32_t max = 0xFFFFFFFF;
    uint32_t part = max / threads;

    for (uint32_t seed = index * part; seed < (index + 1) * part; ++seed) {
        bool found = true;
        for (size_t i = 0; i < 10; ++i) {
            const uint32_t* coeffs = &forward_matrix[((begin + i) - 10) * STATE_SIZE];
            uint32_t sum = coeffs[0] *
                static_cast<uint32_t>(static_cast<int64_t>(static_cast<int32_t>(seed)) * pows[0]);

            for (size_t j = 1; j < STATE_SIZE; ++j) {
                sum += coeffs[j] * 
                    static_cast<uint32_t>((static_cast<int64_t>(seed) * pows[j]) % MODULO);
            }         

            if (TARGET[i] != ALPHABET[static_cast<size_t>(sum >> 1) % 62]) {
                found = false;
                break;
            }
        }
        if (found) {
            printf("found %lu\n", seed);
        }
    }
}

__global__ void
fast_forward(uint32_t* forward_matrix)
{
    size_t blockId = (gridDim.x * blockIdx.y) + blockIdx.x;
    //size_t thread_id = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
    size_t thread_id = threadIdx.x;

    constexpr size_t FORWARD_BY = 100'000'000 + 310;
    constexpr size_t STATE_SIZE = 31;
    constexpr uint64_t M_THREADS = STATE_SIZE*3;

    uint64_t work = FORWARD_BY * STATE_SIZE / M_THREADS;

    //printf("dims: %lu %lu %lu, %lu\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);

    for (int64_t id = thread_id; work-- != 0; id += M_THREADS) {
        int i = id / STATE_SIZE;
        int j = id % STATE_SIZE;

        int start = (3 + i) % STATE_SIZE;
        int finish = (0 + i) % STATE_SIZE;
        forward_matrix[start * STATE_SIZE + j] += forward_matrix[finish * STATE_SIZE + j];
        //printf("%lu : %lu \n", id, thread_id);
        __syncthreads();
    }

}


int
main(void)
{
    int64_t pows[STATE_SIZE] = { 1 };
    for (size_t i = 1; i < STATE_SIZE; ++i) {
        pows[i] = (pows[i - 1] * 16'807) % MODULO;
    }

    uint32_t forward_matrix[STATE_SIZE * STATE_SIZE]{};
    for (size_t i = 0; i < STATE_SIZE; ++i) {
        forward_matrix[i * STATE_SIZE + i] = 1;
    }

    size_t begin = 3;
    size_t end = 0;


    constexpr uint64_t M_THREADS = STATE_SIZE*3;
    uint64_t work = FORWARD_BY * STATE_SIZE / M_THREADS;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;


    // Allocate the device input vector A
    uint32_t* d_forward = NULL;
    err = hipMalloc((void**)&d_forward, STATE_SIZE * STATE_SIZE * sizeof(uint32_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy input data from the host memory to the CUDA device\n");
    //err = hipMemcpy(d_forward, test_matrix, STATE_SIZE * STATE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);
    err = hipMemcpy(d_forward, forward_matrix, STATE_SIZE * STATE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("forwarding");
    fflush(stdout);

    // Launch the Vector Add CUDA Kernel

    auto t1 = std::chrono::system_clock::now();

    fast_forward <<<1, M_THREADS >>> (d_forward);
    hipDeviceSynchronize();

    auto t2 = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = t2 - t1;
    std::cout << diff.count() << " s\n";

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copy output data from the CUDA device to the host memory\n");
    //err = hipMemcpy(test_matrix, d_forward, STATE_SIZE * STATE_SIZE * sizeof(uint32_t), hipMemcpyDeviceToHost);
    err = hipMemcpy(forward_matrix, d_forward, STATE_SIZE * STATE_SIZE * sizeof(uint32_t), hipMemcpyDeviceToHost);


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //cuda leftovers
    for (int64_t id = work* M_THREADS; id < FORWARD_BY * STATE_SIZE; ++id) {
        int i = id / STATE_SIZE;
        int j = id % STATE_SIZE;

        int start = (3 + i) % STATE_SIZE;
        int finish = (0 + i) % STATE_SIZE;
        //test_matrix[start * STATE_SIZE + j] += test_matrix[finish * STATE_SIZE + j];
        forward_matrix[start * STATE_SIZE + j] += forward_matrix[finish * STATE_SIZE + j];
    }

    

    //for (size_t i = 0; i < STATE_SIZE * STATE_SIZE; ++i) {
     //   printf("%lu %lu\n", forward_matrix[i], test_matrix[i]);
      //  if (forward_matrix[i] != test_matrix[i]) {
      //      return -1;
       // }
   // }



    // Allocate the device input vector A
    uint32_t*d_matrix = NULL;
    err = hipMalloc((void **)&d_matrix, STATE_SIZE*STATE_SIZE*sizeof(uint32_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    int64_t*d_pows = NULL;
    err = hipMalloc((void **)&d_pows, STATE_SIZE * sizeof(int64_t));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_matrix, forward_matrix, STATE_SIZE * STATE_SIZE * sizeof(uint32_t), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_pows, pows, STATE_SIZE * sizeof(int64_t), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    begin = 17;
    printf("running %lu\n", begin);
    fflush(stdout);

    // Launch the Vector Add CUDA Kernel

     t1 = std::chrono::system_clock::now();

    bruteforce <<<196, 256>>>(d_matrix, d_pows, begin);

    hipDeviceSynchronize();

     t2 = std::chrono::system_clock::now();
     diff = t2 - t1;
    std::cout << diff.count() << " s\n";


    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_forward);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_matrix);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_pows);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

